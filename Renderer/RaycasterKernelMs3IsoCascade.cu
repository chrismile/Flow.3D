#include "hip/hip_runtime.h"
//#define RAYCASTER_ENABLE_MS3_ISO_CASCADE

#include "RaycasterKernelParams.h"
#ifdef RAYCASTER_ENABLE_MS3_ISO_CASCADE
#include "cudaUtil.h"
#include "RaycasterKernelDefines.h"


#include "RaycasterKernelGlobals.cui"
#include "RaycasterKernelHelpers.cui"

#include "RaycasterKernelMs3IsoCascadeStep.cui"



template < eMeasure M, eTextureFilterMode F, eMeasureComputeMode C, eColorMode CM > 
__global__ void ms3IsoCascadeKernel(
	int2 brickMinScreen,
	int2 brickSizeScreen,
	int2 renderTargetOffset,
	float3 boxMin,
	float3 boxMax,
	float3 world2texOffset,
	float world2texScale
)
{
	const float opacityThreshold = 0.999f;

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= brickSizeScreen.x) || (y >= brickSizeScreen.y)) return;
	x += brickMinScreen.x;
	y += brickMinScreen.y;

	// calculate eye ray in world space
	float3 rayPos = getRayPos(c_raycastParams.viewInv);
	float3 rayDir = getRayDir(c_raycastParams.viewInv, x, y);

	x += renderTargetOffset.x;
	y += renderTargetOffset.y;

	// find intersection with box
	float tnear, tfar;
	if (!intersectBox(rayPos, rayDir, boxMin, boxMax, &tnear, &tfar)) return;
	tnear = fmaxf(tnear, 0.0f); // clamp to near plane

	// get initial color from render target
	uchar4 colorStart;
	surf2Dread(&colorStart, g_surfTarget, x * 4, y);

	float4 sum = rgbaUCharToFloat(colorStart);

	// get rendering params
	float measureScale = c_raycastParams.measureScale;
	float3 isoValues = c_raycastParams.isoValues;
	float4 isoColor1 = c_raycastParams.isoColor1;
	float4 isoColor2 = c_raycastParams.isoColor2;
	float4 isoColor3 = c_raycastParams.isoColor3;

	// march along ray from front to back, accumulating color
	float3 pos = rayPos + rayDir * tnear;
	float3 step = rayDir * c_raycastParams.stepSizeWorld;

	// get value at entry point
	bool bWasInsideCoarseTube = (isoValues.x < getMeasure<M,F,C>(g_texVolume3, w2t(pos), measureScale));
	bool bWasInsideMidTube	  = (isoValues.y < getMeasure<M,F,C>(g_texVolume2, w2t(pos), measureScale));
	pos += step;

	// ray march through volume
	int numSteps = int(ceilf((tfar - tnear) / c_raycastParams.stepSizeWorld)) - 1;
	while(numSteps-- > 0 && sum.w < opacityThreshold) 
	{
		ms3IsoCascadeStep<M,F,C,CM>(sum, bWasInsideCoarseTube, bWasInsideMidTube, world2texOffset, world2texScale, pos, step, rayDir, measureScale, isoValues, isoColor1, isoColor2, isoColor3);
		pos += step;
	}

	// last step at exit point
	pos = rayPos + rayDir * tfar;
	ms3IsoCascadeStep<M,F,C,CM>(sum, bWasInsideCoarseTube, bWasInsideMidTube, world2texOffset, world2texScale, pos, step, rayDir, measureScale, isoValues, isoColor1, isoColor2, isoColor3);

	// write output color
	surf2Dwrite(rgbaFloatToUChar(sum), g_surfTarget, x * 4, y);
}
#endif


void raycasterKernelMs3IsoCascade(RaycasterKernelParams& params)
{
#ifdef RAYCASTER_ENABLE_MS3_ISO_CASCADE
	switch(params.filterMode) {
		#ifdef RAYCASTER_ENABLE_LINEAR
		case TEXTURE_FILTER_LINEAR : RAYCASTER_COLOR_SWITCH(ms3IsoCascadeKernel, TEXTURE_FILTER_LINEAR); break;
		#endif
		#ifdef RAYCASTER_ENABLE_CUBIC
		case TEXTURE_FILTER_CUBIC  : RAYCASTER_COLOR_SWITCH(ms3IsoCascadeKernel, TEXTURE_FILTER_CUBIC); break;
		#endif
	}
#endif
}
