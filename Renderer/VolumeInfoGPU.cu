#include "hip/hip_runtime.h"
#include "VolumeInfoGPU.h"

#include <hip/hip_runtime.h>

#include <TimeVolumeInfo.h>

#include "cudaTum3D.h"
#include "cudaUtil.h"

using namespace tum3D;


__constant__ VolumeInfoGPU c_volumeInfo;
VolumeInfoGPU g_volumeInfo;


void VolumeInfoGPU::Fill(const TimeVolumeInfo& info)
{
	volumeSizeVoxels    = make_uint3(make_int3(info.GetVolumeSize()));
	volumeHalfSizeWorld = make_float3(info.GetVolumeHalfSizeWorld());

	gridSpacing = info.GetGridSpacing();
	timeSpacing = info.GetTimeSpacing();

	brickSizeVoxelsWithOverlap = info.GetBrickSizeWithOverlap();
	brickSizeWorld             = info.GetBrickSizeWorld();
	brickOverlapWorld          = info.GetBrickOverlapWorld();
	brickCount                 = make_uint3(make_int3(info.GetBrickCount()));

	velocityScale = info.GetPhysicalToWorldFactor();
}

void VolumeInfoGPU::Upload(bool cpuTracing) const
{
	if(cpuTracing)
	{
		memcpy(&g_volumeInfo, this, sizeof(g_volumeInfo));
	}
	else
	{
		cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_volumeInfo), this, sizeof(*this), 0, hipMemcpyHostToDevice));
	}
}
