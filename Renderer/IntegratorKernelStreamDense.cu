#include "hip/hip_runtime.h"
#include <global.h>

#include "AdvectMode.h"
#include "BrickIndexGPU.h"
#include "BrickRequestsGPU.h"
#include "IntegrationParamsGPU.h"
#include "LineInfoGPU.h"
#include "TextureFilterMode.h"
#include "TracingCommon.h"
#include "VolumeInfoGPU.h"

#include "AdvectDense.cuh"
#include "Coords.cuh"
#include "IntegratorCommon.cuh"
#include "TextureFilter.cuh"
#include "Jacobian.cuh"

extern __constant__ VolumeInfoGPU c_volumeInfo;
extern __constant__ BrickIndexGPU c_brickIndex;
extern __constant__ BrickRequestsGPU c_brickRequests;
extern __constant__ IntegrationParamsGPU c_integrationParams;
extern __constant__ LineInfoGPU c_lineInfo;

extern texture<float4, hipTextureType3D, hipReadModeElementType> g_texVolume1;


template<eAdvectMode advectMode, eTextureFilterMode filterMode>
__global__ void integrateStreamLinesDenseKernel()
{
	uint lineIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if(lineIndex >= c_lineInfo.lineCount)
		return;

	uint lineLength = c_lineInfo.pVertexCounts[lineIndex];

	if(lineLength >= c_lineInfo.lineLengthMax)
		return;

	LineVertex vertex;
	// get initial position from checkpoints array
	vertex.Position = c_lineInfo.pCheckpoints[lineIndex].Position;
	vertex.Time     = c_lineInfo.pCheckpoints[lineIndex].Time;
	vertex.SeedPosition = c_lineInfo.pCheckpoints[lineIndex].SeedPosition;

	if(vertex.Time >= c_integrationParams.timeMax || c_volumeInfo.isOutsideOfDomain(vertex.Position))
		return;


	// find brick we're in
	float3 brickBoxMin;
	float3 brickBoxMax;
	float3 world2texOffset;
	float3 world2texScale;
	if(!findBrick(vertex.Position, brickBoxMin, brickBoxMax, world2texOffset, world2texScale)) {
		return;
	}

	// get velocity at initial position
	float4 vel4 = sampleVolume<filterMode, float4, float4>(g_texVolume1, w2t(vertex.Position));
	vertex.Velocity = c_volumeInfo.velocityScale * make_float3(vel4.x, vel4.y, vel4.z);

	vertex.LineID = lineIndex;


	// this always points to the next vertex to be written out
	LineVertex* pVertices = c_lineInfo.pVertices + lineIndex * c_lineInfo.vertexStride + lineLength;

	if(lineLength == 0) {
		// new line - build normal: arbitrary vector perpendicular to velocity
		float3 tangent = normalize(vertex.Velocity);
		vertex.Normal = cross(tangent, make_float3(1.0f, 0.0f, 0.0f));
		if(length(vertex.Normal) < 0.01f) vertex.Normal = cross(tangent, make_float3(0.0f, 1.0f, 0.0f));
		vertex.Normal = normalize(vertex.Normal);
		vertex.Jacobian = getJacobian<filterMode>(g_texVolume1, w2t(vertex.Position), c_integrationParams.gridSpacing);
		float3 gradT = sampleScalarGradient<filterMode>(g_texVolume1, w2t(vertex.Position), c_integrationParams.gridSpacing);
		vertex.heat = make_float4(gradT, vel4.w);

		// write out initial vertex
		*pVertices++ = vertex;
		++lineLength;
	} else {
		// existing line - get old normal
		vertex.Normal = c_lineInfo.pCheckpoints[lineIndex].Normal;
	}

	// get the last vertex that was written out
	float3 lastOutPos  = (pVertices - 1)->Position;
	float  lastOutTime = (pVertices - 1)->Time;

	float deltaTime = c_lineInfo.pCheckpoints[lineIndex].DeltaT;

	uint step = 0;
	uint stepsAccepted = 0;

	// dense output
	const uint coeffCount = advectDenseInfo<advectMode>::OutputCoeffCount;
	float3 outputCoeffs[coeffCount];

	bool stayedInAvailableBrick = true;
	while(step < c_integrationParams.stepCountMax &&
		  vertex.Time < c_integrationParams.timeMax &&
		  lineLength < c_lineInfo.lineLengthMax)
	{
		float deltaTimeBak = deltaTime;
		// limit deltaTime ..
		// .. so we don't integrate past timeMax
		deltaTime = min(deltaTime, c_integrationParams.timeMax - vertex.Time);
		// .. so we don't leave the current brick's safe region
		float distMax = c_integrationParams.brickSafeMarginWorld + distanceToBrickBorder(vertex.Position, brickBoxMin, brickBoxMax);
		deltaTime = min(deltaTime, distMax / c_integrationParams.velocityMaxWorld);
		// integrate
		float deltaTimeThisStep = deltaTime;
		bool stepAccepted = advectDense<advectMode, filterMode>(
			g_texVolume1,
			vertex.Position, vertex.Time, vertex.Velocity,
			deltaTime,
			outputCoeffs,
			world2texOffset, world2texScale,
			c_volumeInfo.velocityScale);
		++step;
		if(stepAccepted) {
			++stepsAccepted;

			// if we artificially limited deltaTime earlier, reset it now
			// (if we didn't, the new deltaTime is larger than the backup anyway)
			deltaTime = fmax(deltaTime, deltaTimeBak);

			float3 posDiff = vertex.Position - lastOutPos;
			float timeDiff = vertex.Time     - lastOutTime;
			float posDiffSqr = dot(posDiff, posDiff);
			if((posDiffSqr >= c_integrationParams.outputPosDiffSquared) || (timeDiff >= c_integrationParams.outputTimeDiff)) {
				//get jacobian and heat for measures
				vertex.Jacobian = getJacobian<filterMode>(g_texVolume1, w2t(vertex.Position), c_integrationParams.gridSpacing);
				vel4 = sampleVolume<filterMode, float4, float4>(g_texVolume1, w2t(vertex.Position));
				float3 gradT = sampleScalarGradient<filterMode>(g_texVolume1, w2t(vertex.Position), c_integrationParams.gridSpacing);
				vertex.heat = make_float4(gradT, vel4.w);
				
				// write out interpolated positions
				uint intervalCount = max(1, uint(sqrt(posDiffSqr / c_integrationParams.outputPosDiffSquared)));
				intervalCount = min(intervalCount, c_lineInfo.lineLengthMax - lineLength);
				// interval == 0 corresponds to the old position, interval == intervalCount to the new one
				LineVertex tmpVertex = vertex;
				for(uint interval = 1; interval < intervalCount; ++interval) {
					float3 tmp[coeffCount];
					// position:
					// copy coefficients
					for(uint i = 0; i < coeffCount; ++i) {
						tmp[i] = outputCoeffs[i];
					}
					// evaluate bezier segment using de Casteljau's scheme
					float t = float(interval) / float(intervalCount);
					for(uint l = 1; l < coeffCount; ++l) {
						for(uint i = coeffCount - 1; i >= l; --i) {
							tmp[i] = (1.0f - t) * tmp[i - 1] + t * tmp[i];
						}
					}
					tmpVertex.Position = tmp[coeffCount - 1];
					tmpVertex.Time = vertex.Time - (1.0f - t) * deltaTimeThisStep;
					// velocity:
					for(uint i = 0; i < coeffCount - 1; ++i) {
						tmp[i] = outputCoeffs[i+1] - outputCoeffs[i];
					}
					for(uint l = 1; l < coeffCount - 1; ++l) {
						for(uint i = coeffCount - 2; i >= l; --i) {
							tmp[i] = (1.0f - t) * tmp[i - 1] + t * tmp[i];
						}
					}
					tmpVertex.Velocity = float(coeffCount - 1) * tmp[coeffCount - 2] / deltaTimeThisStep;

					// re-orthogonalize normal wrt. tangent == velocity direction
					float3 binormal = cross(tmpVertex.Velocity, tmpVertex.Normal);
					tmpVertex.Normal = normalize(cross(binormal, tmpVertex.Velocity));
					// and write out the interpolated vertex
					*pVertices++ = tmpVertex;
					++lineLength;
				}


				// re-orthogonalize normal wrt. tangent == velocity direction
				float3 binormal = cross(vertex.Velocity, tmpVertex.Normal);
				vertex.Normal = normalize(cross(binormal, vertex.Velocity));
				// write out final step position
				*pVertices++ = vertex;
				++lineLength;

				lastOutPos  = vertex.Position;
				lastOutTime = vertex.Time;
			} else {
				// even if we don't output anything, we still need to
				// re-orthogonalize normal wrt. tangent == velocity direction
				float3 binormal = cross(vertex.Velocity, vertex.Normal);
				vertex.Normal = normalize(cross(binormal, vertex.Velocity));
			}

			// check if we left the current brick
			if(!isInBrick(vertex.Position, brickBoxMin, brickBoxMax)) {
				bool isOutOfDomain = c_volumeInfo.isOutsideOfDomain(vertex.Position);
				if(isOutOfDomain) {
					// write out final position
					*pVertices++ = vertex;
					++lineLength;

					lastOutPos  = vertex.Position;
					lastOutTime = vertex.Time;
				}
				if(isOutOfDomain || !findBrick(vertex.Position, brickBoxMin, brickBoxMax, world2texOffset, world2texScale)) {
					// new brick isn't available (or we went out of the domain) - get outta here
					// (if we're still inside the domain, the new brick has already been requested in findBrick!)
					stayedInAvailableBrick = false;
					break;
				} else {
					// semi-HACK: update velocity from new brick (can be different to previous one because of lossy compression)
					//            this avoids excessively small time steps at some brick boundaries
					vertex.Velocity = c_volumeInfo.velocityScale * sampleVolume<filterMode, float4, float3>(g_texVolume1, w2t(vertex.Position));
				}
			}
		}
	}


	c_lineInfo.pVertexCounts[lineIndex] = lineLength;
	//assert(c_lineInfo.pVertexCounts[lineIndex] < lineLengthMax);

	// update checkpoint for next integration round
	c_lineInfo.pCheckpoints[lineIndex].Position = vertex.Position;
	c_lineInfo.pCheckpoints[lineIndex].Time     = vertex.Time;
	c_lineInfo.pCheckpoints[lineIndex].Normal   = vertex.Normal;
	c_lineInfo.pCheckpoints[lineIndex].DeltaT   = deltaTime;

	c_lineInfo.pCheckpoints[lineIndex].StepsAccepted += stepsAccepted;
	c_lineInfo.pCheckpoints[lineIndex].StepsTotal    += step;

	// if the line is still alive and in an available brick, request it again for next round
	if(vertex.Time < c_integrationParams.timeMax &&
	   lineLength < c_lineInfo.lineLengthMax &&
	   stayedInAvailableBrick)
	{
		// find out which brick we're in now
		uint3 brickIndex = c_volumeInfo.getBrickIndex(vertex.Position);
		uint brickLinearIndex = c_volumeInfo.getBrickLinearIndex(brickIndex);
		// request it to be loaded
		c_brickRequests.requestBrick(brickLinearIndex);
	}
}


#include "cudaUtil.h"

#include "IntegratorKernelDefines.h"


void integratorKernelStreamLinesDense(const LineInfo& lineInfo, eAdvectMode advectMode, eTextureFilterMode filterMode)
{
	uint blockSize = 128; //TODO try out different values
	uint blockCount = (lineInfo.lineCount + blockSize - 1) / blockSize;

#define INTEGRATE(advect, filter) integrateStreamLinesDenseKernel <advect, filter> <<<blockCount, blockSize>>> ()

	ADVECT_DENSE_SWITCH;
	cudaCheckMsg("integrateStreamLinesDenseKernel execution failed");

#undef INTEGRATE
}
