#include "IntegratorTimeInCell.cuh"

#include <iostream>

texture<uint32, hipTextureType2D, hipReadModeElementType> g_cellTexture;

void IntegratorTimeInCell::Upload(CellTextureGPU& info, uint32 * textureMemCPU, size_t width, size_t height)
{
	Free(info);

	cudaSafeCall(hipMallocArray(&info.textureArray, &g_cellTexture.channelDesc, width, height));
	cudaSafeCall(hipMemcpyToArray(info.textureArray, 0, 0, textureMemCPU, width * height * sizeof(uint32), hipMemcpyHostToDevice));
	hipChannelFormatDesc channelFormat = { 32, 0, 0, 0, hipChannelFormatKindUnsigned };
	hipBindTextureToArray(g_cellTexture, info.textureArray);
	g_cellTexture.normalized = true;
	g_cellTexture.addressMode[0] = hipAddressModeClamp;

	std::cout << "IntegratorTimeInCell: CellTexture uploaded" << std::endl;
}

void IntegratorTimeInCell::Free(CellTextureGPU & info)
{
	if (info.textureArray == NULL) return;
	cudaSafeCall(hipUnbindTexture(g_cellTexture));
	cudaSafeCall(hipFreeArray(info.textureArray));
	info.textureArray = NULL;
	std::cout << "IntegratorTimeInCell: CellTexture freed" << std::endl;
}

__device__ void IntegratorTimeInCell::processParticle(LineVertex * vertex, float deltaTime)
{
	
}
