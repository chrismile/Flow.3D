#include "LineInfoGPU.h"

#include <hip/hip_runtime.h>
#include <iostream>

#include "cudaUtil.h"


//__constant__ LineInfoGPU c_lineInfo;
LineInfoGPU g_lineInfo;


void LineInfoGPU::Upload(bool cpuTracing) const
{
	std::cout << "LineInfoGPU::Upload" << std::endl;

	if(cpuTracing)
		memcpy(&g_lineInfo, this, sizeof(g_lineInfo));
	//else
		//cudaSafeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_lineInfo), this, sizeof(*this), 0, hipMemcpyHostToDevice));
}
